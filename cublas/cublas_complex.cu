#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	if (abort) exit(code);
	}
}

void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ?"Yes" : "No"));
    return;
}

//__device__ static hipDoubleComplex alpha_gpu;  
//__device__ static hipDoubleComplex beta_gpu; 


struct ZGEMVScalarParams {
    hipDoubleComplex alpha, beta;
};

void call_zgemv(hipblasHandle_t handle, int m, int n, hipDoubleComplex alpha_gpu, hipDoubleComplex *A_gpu, int ndim, hipDoubleComplex* vec_gpu, hipDoubleComplex beta_gpu, hipDoubleComplex* result_gpu) {
        hipblasZgemv(handle, HIPBLAS_OP_N, m, n, &alpha_gpu, A_gpu, ndim, vec_gpu, 1, &beta_gpu, result_gpu, 1);
}

__global__ void call_zgemv_gpu(int m, int n, hipDoubleComplex* alpha_gpu, hipDoubleComplex *A_gpu, int ndim, hipDoubleComplex* vec_gpu, hipDoubleComplex* beta_gpu, hipDoubleComplex* result_gpu) {

	hipblasHandle_t handle; 
	hipblasCreate(&handle);
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid == 0) {
	        hipblasZgemv(handle, HIPBLAS_OP_N, m, n, alpha_gpu, A_gpu, ndim, vec_gpu, 1, beta_gpu, result_gpu, 1);
		hipDeviceSynchronize();
	}
	hipblasDestroy(handle);
}




void print_double_complex_matrix(hipDoubleComplex matrix[], int N) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("%f + %f j  \t", hipCreal(matrix[i*N + j]), hipCimag(matrix[i*N + j]));
		}
		printf("\n");
	}
}

void print_double_complex_vector(hipDoubleComplex vector[], int N) {
	for (int i = 0; i < N; i++) {
		printf("%f + %f j \t", hipCreal(vector[i]), hipCimag(vector[i]));
	}
	printf("\n");
}




int main(int argc, char* argv[]) {

        // Initial Machinery to select the GPU

        hipDeviceProp_t prop; // Blank
        memset(&prop, 0, sizeof(hipDeviceProp_t)); // Set struct to all 0

        int devcount;
        gpuErrchk(hipGetDeviceCount(&devcount));
        printf("Devcount %d! \n", devcount);

        // Retrieve GPU properties - Look for a K20 or above    
        int dev;
        prop.multiProcessorCount = 13;
        hipChooseDevice(&dev, &prop);
        hipGetDeviceProperties(&prop, dev);


        printf(" *** DEVICE INFORMATION IS AS FOLLOWS *** \n");
        printDevProp(prop);
        printf(" *** END DEVICE INFORMATION  *** \n");





	
	// Instantiate mock data

	if (argc < 2) {
		printf("Please enter a value for ndim. \n");
		exit(1);
	}
	if (argc > 2) {
		printf("Please enter ONLY one value: ndim \n");
		exit(1);
	}

	int ndim = atoi(argv[1]);
	printf("Testing cuBLAS for a %d dimensional matrix... \n", ndim);


	hipDoubleComplex* A = (hipDoubleComplex *)malloc(ndim*ndim*sizeof(hipDoubleComplex)); 
	for (int i = 0; i < ndim; i++) {
		for (int j = 0; j < ndim; j++) {
			if (i == j) {
				A[ndim * i + j] = make_hipDoubleComplex(i, i); 	
			}
			else {
				A[ndim * i + j] = make_hipDoubleComplex(0, 0);
			}
		}
	}

	print_double_complex_matrix(A, ndim);

	printf("Which we multiply into the vector... \n");

	hipDoubleComplex* vec = (hipDoubleComplex *)malloc(ndim*sizeof(hipDoubleComplex));

	for (int i = 0; i < ndim; i++) {
		vec[i] = make_hipDoubleComplex(1.0, 1.0);
	}

	print_double_complex_vector(vec, ndim);

	// Allocate memory on the gpu

	printf("Copying memory down on to the cuda device...\n");

	hipDoubleComplex* A_gpu;
	hipDoubleComplex* vec_gpu;

	int nblas = 1;

	// CAN WE CALL CUBLAS ALL ON THE SAME MATRIX? TRY CALLING IT 10 TIMES	

	hipDoubleComplex* result = (hipDoubleComplex *)malloc(nblas*ndim * sizeof(hipDoubleComplex));
	hipDoubleComplex* result_gpu;

	if ( hipSuccess != hipMalloc((void**)&A_gpu, ndim*ndim*sizeof(hipDoubleComplex)) ) {
                printf("hipMalloc Failed...\n");
                exit(1);
        }
	if ( hipSuccess != hipMalloc((void**)&vec_gpu, ndim*sizeof(hipDoubleComplex)) ) {
                printf("hipMalloc Failed...\n");
                exit(1);
        }
	// WE NOW NEED 10X THE MEMORY ALLOCATED FOR RESULTS
	if ( hipSuccess != hipMalloc((void**)&result_gpu, nblas*ndim*sizeof(hipDoubleComplex)) ) {
                printf("hipMalloc Failed...\n");
                exit(1);
        }

	hipMemcpy(A_gpu, A, ndim*ndim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
	hipMemcpy(vec_gpu, vec, ndim*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);


	// Initialize cublas

	int m = ndim;// Square Matrix
	int n = ndim;// Square Matrix
	dim3 grid = dim3(1, 1, 1);
	dim3 block = dim3(nblas, 1, 1);


//	hipblasHandle_t handle; 
//	hipblasCreate(&handle);

	ZGEMVScalarParams h_scalar_params = {make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)}; 
	ZGEMVScalarParams *d_scalar_params;
	

	if (hipSuccess != hipMalloc((void **)&d_scalar_params, sizeof(ZGEMVScalarParams)) ) {    
		printf("CudaMalloc Failed... \n");
		exit(1);
   	}

	if (hipSuccess != hipMemcpy(d_scalar_params, &h_scalar_params, sizeof(ZGEMVScalarParams), hipMemcpyHostToDevice) ) {
        	printf("!!!! host to device memory copy error\n");
        	exit(1);
    	}



//      hipblasZgemv(handle, HIPBLAS_OP_N, m, n, &alpha_gpu, A_gpu, ndim, vec_gpu, 1, &beta_gpu, result_gpu, 1);

        call_zgemv_gpu<<<1,1>>>(m, n, &d_scalar_params->alpha, A_gpu, ndim, vec_gpu, &d_scalar_params->beta, result_gpu);

//	call_zgemv(handle, m, n, alpha_gpu, A_gpu, ndim, vec_gpu, beta_gpu, result_gpu);



	printf("Copying results up to host... \n");
	hipMemcpy(result, result_gpu, nblas*ndim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
	printf("Result is vector \n");
	print_double_complex_vector(result, ndim);
//	print_double_complex_vector(&result[ndim], ndim);
	hipFree(result_gpu);
	hipFree(A);
	hipFree(vec);
	hipFree(d_scalar_params);
	free(result);
	free(vec);
	free(A);

	
	return 0;
}
